#include "nn.h"


vector<float> one_hot(vector<int> y, int n_samples, int n_classes) {
    vector<float> onehots(n_samples * n_classes, 0);
    for (int i = 0; i < y.size(); i++) {
        int idx = n_classes * i + y.at(i);
        onehots[idx] = 1.0;
    }

    return onehots;
}


float loss(float* y_pred, float* y_true, int n_samples, int n_classes) {
    float* abs_err = _add_CPU(y_pred, y_true, n_samples * n_classes, -1);
    float cee = _sum_CPU(abs_err, n_samples * n_classes);
    return (-1 * cee) / n_samples;
}


vector<float*> forward(vector<float> X, vector<vector<float>> Ws, int n_samples, int n_features, 
                        int hidden_size, int out_size, bool use_gpu, bool optimize) {

    vector<float*> outs;
    if (use_gpu) { 
        if (optimize) {
            outs = _fw_GPU_optim(X, Ws, n_samples, n_features, hidden_size, out_size);

            // Set first output as input data
            outs[0] = X.data();
            // cout << "in:" << outs.at(0)[0] << endl;
        } else 
            outs = _fw_GPU(X, Ws, n_samples, n_features, hidden_size, out_size);

            // Set first output as input data
            outs[0] = X.data();
            // cout << "in:" << outs.at(0)[0] << endl;
    } else {
        //-- Forward using CPU

        outs.push_back(X.data());

        int layer_in_size = n_features;
        int layer_out_size = hidden_size;
        for (int i = 0; i < Ws.size(); i++) {
            if (i != 0) layer_in_size = hidden_size;
            if (i == Ws.size() - 1) layer_out_size = out_size;
    
            vector<float> W = Ws[i];
            float* X_in = outs[i];

            // Multiply
            float* out = _matmul_CPU(X_in, W.data(), n_samples, layer_in_size, layer_out_size);

            // Activation function
            if (i == Ws.size() - 1)
                out = _softmax_CPU(out, n_samples, out_size);
            else
                out = _ReLU_CPU(out, n_samples * hidden_size);

            outs.push_back(out);
        }
    }

    return outs;
}


vector<float*> backward(vector<float*> outs, vector<vector<float>> Ws, 
                        vector<float> y_onehot, int n_samples, int n_features, 
                        int hidden_size, int n_classes) {

    vector<float*> gradients(Ws.size());

    // Final output layer error
    // delta_out = final_output - y_onehot
    float* final_output = outs.back();
    float* delta_out = _add_CPU(final_output, y_onehot.data(), n_samples * n_classes, -1); 

    // Final layer gradient
    // TODO: divide grad_out by n_samples
    float* final_input = outs[outs.size() - 2];  // Input to the final layer
    float* final_input_T = _transpose(final_input, n_samples, hidden_size);
    float* grad_out = _matmul_CPU(final_input_T, delta_out, hidden_size, n_samples, n_classes);

    // Store gradient
    gradients.back() = grad_out; 

    free(final_input_T);
    free(grad_out);

    // BEGIN BACKPROPAGATION
    float* delta_hidden = delta_out;
    int layer_input_size = hidden_size;
    int layer_output_size = hidden_size;
    for (int layer = Ws.size() - 2; layer > -1; layer--) {

        if (layer == 0) layer_input_size = n_features;
        cout << "layer: " << layer << endl;

        // Current layer input + outputs
        float* layer_input = outs[layer];
        float* layer_output = outs[layer + 1];

        // Obtain next layer's weights, input + output sizes
        int next_layer = layer + 1;
        vector<float> W_next = Ws[next_layer];
        int next_layer_input_size = layer_output_size;
        int next_layer_output_size = hidden_size;
        if (next_layer == Ws.size() - 1) next_layer_output_size = n_classes;

        // ReLU derivative
        float* dReLU = _dReLU_CPU(layer_output, n_samples * layer_output_size);

        // Transpose next layer's weights
        float* W_next_T = _transpose(W_next.data(), next_layer_input_size, next_layer_output_size);

        // Current layer's output error
        float* delta_hidden_temp = _matmul_CPU(delta_hidden, W_next_T, n_samples, next_layer_output_size, next_layer_input_size);
        float* delta_hidden_new = _ewmul_CPU(delta_hidden_temp, dReLU, n_samples * layer_output_size);

        free(delta_hidden);
        free(delta_hidden_temp);
        free(dReLU);

        // Update output error
        delta_hidden = delta_hidden_new;

        // TODO: divide grad_hidden by n_samples
        float* layer_input_T = _transpose(layer_input, n_samples, layer_input_size);
        float* grad_hidden = _matmul_CPU(layer_input_T, delta_hidden, layer_input_size, n_samples, layer_output_size);

        gradients[layer] = grad_hidden; // Store gradient
        
        free(layer_input_T);
        free(W_next_T);
    }

    free(delta_hidden); // Free memory for the last delta
    return gradients;
}


void train(vector<vector<float>> X, vector<int> y, vector<vector<float>> &Ws,
           int hidden_size, int n_classes, int max_epoch, float learning_rate, bool use_gpu) {
    
    int sample_size = X.size();
    int n_data_features = X.at(0).size();

    // One-hot encoding
    vector<float> y_onehot = one_hot(y, sample_size, n_classes);

    // Flatten input data
    vector<float> X_train(sample_size * n_data_features);
    for (int i = 0; i < sample_size; ++i)
        copy(X[i].begin(), X[i].end(), X_train.begin() + i * n_data_features);

    for (int epoch = 0; epoch < max_epoch; epoch++) {
        // Forward
        vector<float*> outs = forward(X_train, Ws, sample_size, n_data_features, hidden_size, n_classes, use_gpu, false);
        cout << outs.size() << endl;
        cout << "EPOCH " << epoch << " calc grads...\n";
        vector<float*> grads = backward(outs, Ws, y_onehot, sample_size, n_data_features, hidden_size, n_classes);
    }
}


void forwardCUDA(const float* h_X, const float* h_W1, const float* h_b1, 
                 const float* h_W2, const float* h_b2,
                 const float* h_W3, const float* h_b3,
                 float* h_output, int batch_size) {

    const int input_size = 784;
    const int hidden1_size = 128;
    const int hidden2_size = 128;
    const int output_size = 10;

    float *d_X, *d_W1, *d_b1, *d_Z1, *d_A1;
    float *d_W2, *d_b2, *d_Z2, *d_A2;
    float *d_W3, *d_b3, *d_Z3, *d_output;

    size_t size_X = batch_size * input_size * sizeof(float);
    size_t size_hidden1 = batch_size * hidden1_size * sizeof(float);
    size_t size_hidden2 = batch_size * hidden2_size * sizeof(float);
    size_t size_output = batch_size * output_size * sizeof(float);

    hipMalloc(&d_X, size_X);
    hipMalloc(&d_W1, input_size * hidden1_size * sizeof(float));
    hipMalloc(&d_b1, hidden1_size * sizeof(float));
    hipMalloc(&d_Z1, size_hidden1);
    hipMalloc(&d_A1, size_hidden1);

    hipMalloc(&d_W2, hidden1_size * hidden2_size * sizeof(float));
    hipMalloc(&d_b2, hidden2_size * sizeof(float));
    hipMalloc(&d_Z2, size_hidden2);
    hipMalloc(&d_A2, size_hidden2);

    hipMalloc(&d_W3, hidden2_size * output_size * sizeof(float));
    hipMalloc(&d_b3, output_size * sizeof(float));
    hipMalloc(&d_Z3, size_output);
    hipMalloc(&d_output, size_output);

    hipMemcpy(d_X, h_X, size_X, hipMemcpyHostToDevice);
    hipMemcpy(d_W1, h_W1, input_size * hidden1_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, h_b1, hidden1_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, h_W2, hidden1_size * hidden2_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, h_b2, hidden2_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W3, h_W3, hidden2_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b3, h_b3, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Block và Grid size
    dim3 blockSize(16, 16);
    dim3 gridSize1((hidden1_size + blockSize.x - 1) / blockSize.x,
                   (batch_size + blockSize.y - 1) / blockSize.y);

    // Lớp ẩn thứ nhất: Z1 = X * W1 + b1
    _matmul_GPU<<<gridSize1, blockSize>>>(d_X, d_W1, d_Z1, batch_size, input_size, hidden1_size);

    // Thêm bias vào Z1 và áp dụng ReLU
    dim3 block1D((batch_size * hidden1_size + 255) / 256);
    hipMemcpy(d_Z1, d_b1, hidden1_size * sizeof(float), hipMemcpyDeviceToDevice);
    _ReLU_GPU<<<block1D, 256>>>(d_Z1, batch_size * hidden1_size);

    // Lớp ẩn thứ hai: Z2 = Z1 * W2 + b2
    dim3 gridSize2((hidden2_size + blockSize.x - 1) / blockSize.x,
                   (batch_size + blockSize.y - 1) / blockSize.y);
    _matmul_GPU<<<gridSize2, blockSize>>>(d_Z1, d_W2, d_Z2, batch_size, hidden1_size, hidden2_size);

    // Thêm bias vào Z2 và áp dụng ReLU
    dim3 block2D((batch_size * hidden2_size + 255) / 256);
    hipMemcpy(d_Z2, d_b2, hidden2_size * sizeof(float), hipMemcpyDeviceToDevice);
    _ReLU_GPU<<<block2D, 256>>>(d_Z2, batch_size * hidden2_size);

    // Lớp đầu ra: output = Z2 * W3 + b3
    dim3 gridSize3((output_size + blockSize.x - 1) / blockSize.x,
                   (batch_size + blockSize.y - 1) / blockSize.y);
    _matmul_GPU<<<gridSize3, blockSize>>>(d_Z2, d_W3, d_output, batch_size, hidden2_size, output_size);
    dim3 blockSoftmax(256);
    dim3 gridSoftmax((batch_size * output_size + blockSoftmax.x - 1) / blockSoftmax.x);
    _softmax_GPU<<<gridSoftmax, blockSoftmax>>>(d_output, d_output, batch_size, output_size);

    // Copy kết quả từ GPU về CPU
    hipMemcpy(h_output, d_output, size_output, hipMemcpyDeviceToHost);

    // Giải phóng bộ nhớ
    hipFree(d_X);
    hipFree(d_W1);
    hipFree(d_b1);
    hipFree(d_Z1);
    hipFree(d_A1);
    hipFree(d_W2);
    hipFree(d_b2);
    hipFree(d_Z2);
    hipFree(d_A2);
    hipFree(d_W3);
    hipFree(d_b3);
    hipFree(d_Z3);
    hipFree(d_output);
}

// void foward( float* X, float* W1, float* W2, float* W3, float* b1, float* b2, float* b3,
//             int n_input){

    
//     vector<vector<float>> Z1(n_input, 128);
//     vector<vector<float>> A1(n_input,128);
//     vector<vector<float>> Z2(n_input,128);
//     vector<vector<float>> A2(n_input,128);
//     vector<vector<float>> Z3(n_input,10);
//     vector<vector<float>> A3(n_input,1);
//    	dim3 blockSize(32, 32); // Default


//     matMul(X, W1, Z1, n_input, 784,128, blockSize);
//     A1 = ReLU(Z1, n_input);
//     matMul(A1, W2, Z2, n_input, 128,128, blockSize);
//     A2 = ReLU(Z2, n_input);
//     matMul(A2,W3, Z3, n_input, 128,10, blockSize);
//     A3 = softmax(Z3, n_input);

//     return 0;
// }