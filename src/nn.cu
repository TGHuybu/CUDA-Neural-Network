#include "hip/hip_runtime.h"
#include "nn.h"


vector<float> one_hot(vector<int> y, int n_samples, int n_classes) {
    vector<float> onehots(n_samples * n_classes, 0);
    for (int i = 0; i < y.size(); i++) {
        int idx = n_classes * i + y.at(i);
        onehots[idx] = 1.0;
    }

    return onehots;
}


float loss(float* y_pred, float* y_true, int n_samples, int n_classes) {
    // log(y_pred)
    float* log_y_pred = new float[n_samples * n_classes];
    for (int i = 0; i < n_samples * n_classes; i++) {
        log_y_pred[i] = log(fmax(y_pred[i], 1e-7));
    }

    float* temp = _ewmul_CPU(log_y_pred, y_true, n_samples * n_classes);
    float cee = _sum_CPU(temp, n_samples * n_classes);
    return (-1.0 * cee) / n_samples;
}


vector<float*> forward(vector<float> X, vector<vector<float>> Ws, int n_samples, int n_features, 
                        int hidden_size, int out_size, bool use_gpu, bool optimize) {

    vector<float*> outs;
    if (use_gpu) { 
        if (optimize) {
            outs = _fw_GPU_optim(X, Ws, n_samples, n_features, hidden_size, out_size);

            // Set first output as input data
            outs[0] = X.data();
        } else 
            outs = _fw_GPU(X, Ws, n_samples, n_features, hidden_size, out_size);

            // Set first output as input data
            outs[0] = X.data();
    } else {
        //-- Forward using CPU
        outs.push_back(X.data());

        GpuTimer timer;
        float time;

        int layer_in_size = n_features;
        int layer_out_size = hidden_size;
        for (int i = 0; i < Ws.size(); i++) {
            if (i != 0) layer_in_size = hidden_size;
            if (i == Ws.size() - 1) layer_out_size = out_size;

            timer.Start();
    
            vector<float> W = Ws[i];
            float* X_in = outs[i];

            // Multiply
            float* out = _matmul_CPU(X_in, W.data(), n_samples, layer_in_size, layer_out_size);

            // Activation function
            if (i == Ws.size() - 1)
                out = _softmax_CPU(out, n_samples, out_size);
            else
                out = _ReLU_CPU(out, n_samples * hidden_size);

            timer.Stop();
            time = timer.Elapsed();
            cout << "- layer " << i << " ";
            printf("forward time: %f ms\n", time);

            outs.push_back(out);
        }

        outs[0] = X.data();
    }

    return outs;
}


void update_weights(vector<vector<float>> &Ws, vector<float*> gradients, 
                    float learning_rate) {

    for (int i = 0; i < Ws.size(); i++) {
        for (int j = 0; j < Ws[i].size(); j++)
            Ws[i][j] -= learning_rate * gradients[i][j];
    }
}


void train(vector<vector<float>> X, vector<int> y, vector<vector<float>> &Ws,
           int hidden_size, int n_classes, int max_epoch, float learning_rate, bool use_gpu, bool optimize) {
    
    int sample_size = X.size();
    int n_data_features = X.at(0).size();

    // One-hot encoding
    vector<float> y_onehot = one_hot(y, sample_size, n_classes);

    // Flatten input data
    vector<float> X_train(sample_size * n_data_features);
    for (int i = 0; i < sample_size; ++i)
        copy(X[i].begin(), X[i].end(), X_train.begin() + i * n_data_features);

    for (int epoch = 0; epoch < max_epoch; epoch++) {
        // Forward
        vector<float*> outs = forward(X_train, Ws, sample_size, n_data_features, hidden_size, n_classes, use_gpu, true);

        // TODO: Branch out to CPU and GPU backward functions
        vector<float*> grads = _backward_CPU(outs, Ws, y_onehot, sample_size, n_data_features, hidden_size, n_classes);

        // Update weights
        update_weights(Ws, grads, learning_rate);

        float cee = loss(outs.back(), y_onehot.data(), sample_size, n_classes);
        cout << ">>> Epoch " << epoch + 1 << " CEE loss: " << cee << endl;
    }
}