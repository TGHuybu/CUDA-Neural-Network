#include "nn.h"


vector<float*> forward(vector<float> X, vector<vector<float>> Ws, 
                        int n_samples, int n_features, int hidden_size, int out_size, bool use_gpu) {
    
    // Convert to pointer for ease of handling
    float *X_in = X.data();
    
    // Save outputs of each layer
    vector<float*> outs;
    outs.push_back(X_in);

    for (int i = 0; i < Ws.size(); i++) {
        if (i != 0) n_features = hidden_size;
        if (i == Ws.size() - 1) hidden_size = out_size;

        vector<float> W = Ws[i];
        float *out = new float[n_samples * hidden_size];
        cout << n_samples * hidden_size << endl;

        if (use_gpu) {
            // Allocate memory on device
            float *d_X, *d_W, *d_out;
            CHECK(hipMalloc(&d_X, n_samples * n_features *sizeof(float)));
            CHECK(hipMalloc(&d_W, W.size() * sizeof(float)));
            CHECK(hipMalloc(&d_out, n_samples * hidden_size * sizeof(float)));

            // Copy memory: host-to-device
            CHECK(hipMemcpy(d_X, X_in, n_samples * n_features * sizeof(float), hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_W, W.data(), W.size() * sizeof(float), hipMemcpyHostToDevice));

            // Define block and grid size
            dim3 blockSize(16, 16);
            dim3 gridSize((hidden_size + blockSize.x - 1) / blockSize.x,
                            (n_samples + blockSize.y - 1) / blockSize.y);

            // Multiply
            matMul<<<gridSize, blockSize>>>(d_X, d_W, d_out, n_samples, n_features, hidden_size);

            // Activation function
            dim3 blockSize_1D(256);
            dim3 gridSize_1D((n_samples * hidden_size + blockSize_1D.x - 1) / 256);
            if (i == Ws.size() - 1)
                softmax<<<gridSize_1D, blockSize_1D>>>(d_out, d_out, n_samples, out_size);
            else
                ReLU<<<gridSize_1D, blockSize_1D>>>(d_out, n_samples * hidden_size);

            // Copy memory: device-to-host
            CHECK(hipMemcpy(out, d_out, n_samples * hidden_size * sizeof(float), hipMemcpyDeviceToHost));

            // Free device memory
            CHECK(hipFree(d_X));
            CHECK(hipFree(d_W));
            CHECK(hipFree(d_out));

        } else {
            // TODO: CPU forward
        }

        outs.push_back(out);
        X_in = out;
    }

    return outs;
}


void forwardCUDA(const float* h_X, const float* h_W1, const float* h_b1, 
                 const float* h_W2, const float* h_b2,
                 const float* h_W3, const float* h_b3,
                 float* h_output, int batch_size) {

    const int input_size = 784;
    const int hidden1_size = 128;
    const int hidden2_size = 128;
    const int output_size = 10;

    float *d_X, *d_W1, *d_b1, *d_Z1, *d_A1;
    float *d_W2, *d_b2, *d_Z2, *d_A2;
    float *d_W3, *d_b3, *d_Z3, *d_output;

    size_t size_X = batch_size * input_size * sizeof(float);
    size_t size_hidden1 = batch_size * hidden1_size * sizeof(float);
    size_t size_hidden2 = batch_size * hidden2_size * sizeof(float);
    size_t size_output = batch_size * output_size * sizeof(float);

    hipMalloc(&d_X, size_X);
    hipMalloc(&d_W1, input_size * hidden1_size * sizeof(float));
    hipMalloc(&d_b1, hidden1_size * sizeof(float));
    hipMalloc(&d_Z1, size_hidden1);
    hipMalloc(&d_A1, size_hidden1);

    hipMalloc(&d_W2, hidden1_size * hidden2_size * sizeof(float));
    hipMalloc(&d_b2, hidden2_size * sizeof(float));
    hipMalloc(&d_Z2, size_hidden2);
    hipMalloc(&d_A2, size_hidden2);

    hipMalloc(&d_W3, hidden2_size * output_size * sizeof(float));
    hipMalloc(&d_b3, output_size * sizeof(float));
    hipMalloc(&d_Z3, size_output);
    hipMalloc(&d_output, size_output);

    hipMemcpy(d_X, h_X, size_X, hipMemcpyHostToDevice);
    hipMemcpy(d_W1, h_W1, input_size * hidden1_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, h_b1, hidden1_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, h_W2, hidden1_size * hidden2_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, h_b2, hidden2_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W3, h_W3, hidden2_size * output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b3, h_b3, output_size * sizeof(float), hipMemcpyHostToDevice);

    // Block và Grid size
    dim3 blockSize(16, 16);
    dim3 gridSize1((hidden1_size + blockSize.x - 1) / blockSize.x,
                   (batch_size + blockSize.y - 1) / blockSize.y);

    // Lớp ẩn thứ nhất: Z1 = X * W1 + b1
    matMul<<<gridSize1, blockSize>>>(d_X, d_W1, d_Z1, batch_size, input_size, hidden1_size);

    // Thêm bias vào Z1 và áp dụng ReLU
    dim3 block1D((batch_size * hidden1_size + 255) / 256);
    hipMemcpy(d_Z1, d_b1, hidden1_size * sizeof(float), hipMemcpyDeviceToDevice);
    ReLU<<<block1D, 256>>>(d_Z1, batch_size * hidden1_size);

    // Lớp ẩn thứ hai: Z2 = Z1 * W2 + b2
    dim3 gridSize2((hidden2_size + blockSize.x - 1) / blockSize.x,
                   (batch_size + blockSize.y - 1) / blockSize.y);
    matMul<<<gridSize2, blockSize>>>(d_Z1, d_W2, d_Z2, batch_size, hidden1_size, hidden2_size);

    // Thêm bias vào Z2 và áp dụng ReLU
    dim3 block2D((batch_size * hidden2_size + 255) / 256);
    hipMemcpy(d_Z2, d_b2, hidden2_size * sizeof(float), hipMemcpyDeviceToDevice);
    ReLU<<<block2D, 256>>>(d_Z2, batch_size * hidden2_size);

    // Lớp đầu ra: output = Z2 * W3 + b3
    dim3 gridSize3((output_size + blockSize.x - 1) / blockSize.x,
                   (batch_size + blockSize.y - 1) / blockSize.y);
    matMul<<<gridSize3, blockSize>>>(d_Z2, d_W3, d_output, batch_size, hidden2_size, output_size);
    dim3 blockSoftmax(256);
    dim3 gridSoftmax((batch_size * output_size + blockSoftmax.x - 1) / blockSoftmax.x);
    softmax<<<gridSoftmax, blockSoftmax>>>(d_output, d_output, batch_size, output_size);

    // Copy kết quả từ GPU về CPU
    hipMemcpy(h_output, d_output, size_output, hipMemcpyDeviceToHost);

    // Giải phóng bộ nhớ
    hipFree(d_X);
    hipFree(d_W1);
    hipFree(d_b1);
    hipFree(d_Z1);
    hipFree(d_A1);
    hipFree(d_W2);
    hipFree(d_b2);
    hipFree(d_Z2);
    hipFree(d_A2);
    hipFree(d_W3);
    hipFree(d_b3);
    hipFree(d_Z3);
    hipFree(d_output);
}

// void foward( float* X, float* W1, float* W2, float* W3, float* b1, float* b2, float* b3,
//             int n_input){

    
//     vector<vector<float>> Z1(n_input, 128);
//     vector<vector<float>> A1(n_input,128);
//     vector<vector<float>> Z2(n_input,128);
//     vector<vector<float>> A2(n_input,128);
//     vector<vector<float>> Z3(n_input,10);
//     vector<vector<float>> A3(n_input,1);
//    	dim3 blockSize(32, 32); // Default


//     matMul(X, W1, Z1, n_input, 784,128, blockSize);
//     A1 = ReLU(Z1, n_input);
//     matMul(A1, W2, Z2, n_input, 128,128, blockSize);
//     A2 = ReLU(Z2, n_input);
//     matMul(A2,W3, Z3, n_input, 128,10, blockSize);
//     A3 = softmax(Z3, n_input);

//     return 0;
// }