#include "hip/hip_runtime.h"
#include "utils_device.h"


void device_info() {
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");
}

__global__ void _matmul_GPU(float* A, float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    	if (row < m && col < k)
    	{
        	float value = 0;
        	for (int i = 0; i < n; i++) 
        	{
            		value += A[row * n + i] * B[i * k + col];
        	}
        	C[row * k + col] = value; 
    	}
}

// void matMul(float* A, float* B, float* C, int m, int n, int k, dim3 blockSize = dim3(1)){

//         float* d_A, * d_B, * d_C;
// 	CHECK(hipMalloc((void**)&d_A, m * n * sizeof(float)));
// 	CHECK(hipMalloc((void**)&d_B, n * k * sizeof(float)));
// 	CHECK(hipMalloc((void**)&d_C, m * k * sizeof(float)));

//         // TODO: Copy data to device memories
//         CHECK(hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice));
// 	CHECK(hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice));
//         dim3 gridSize((k + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
//  // TODO: Compute gridSize
        
		
// 		matMulkernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

//         // TODO: Copy result from device memory
// 	CHECK(hipMemcpy(C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost));
//         // TODO: Free device memories
// 	CHECK(hipFree(d_A));
// 	CHECK(hipFree(d_B));
// 	CHECK(hipFree(d_C));
	
// 		printf("Grid size: %d * %d, block size: %d * %d\n", 
// 			gridSize.x,gridSize.y, blockSize.x,blockSize.y);

// }

__global__ void _ReLU_GPU(float* Z, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        Z[idx] = fmaxf(0.0f, Z[idx]);
    }
}

// __global__ void softmax(float *input, float *output, int batch_size, int output_size) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int batch_idx = idx / output_size;
//     int output_idx = idx % output_size;

//     if (batch_idx >= batch_size) return;

//     // Shared memory for stable softmax
//     __shared__ float max_val[1024];
//     __shared__ float sum[1024];

//     // Find maximum value in the row
//     float local_max = -FLT_MAX;
//     for (int i = 0; i < output_size; ++i) {
//         local_max = max(local_max, input[batch_idx * output_size + i]);
//     }
//     max_val[threadIdx.x] = local_max;
//     __syncthreads();

//     // Calculate exp(input - max)
//     float exp_sum = 0.0f;
//     for (int i = 0; i < output_size; ++i) {
//         exp_sum += exp(input[batch_idx * output_size + i] - max_val[threadIdx.x]);
//     }
//     sum[threadIdx.x] = exp_sum;
//     __syncthreads();

//     // Normalize
//     output[idx] = exp(input[idx] - max_val[threadIdx.x]) / sum[threadIdx.x];
// }

__global__ void _softmax_GPU(float *input, float *output, int batch_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_idx = idx / output_size;
    int output_idx = idx % output_size;

    if (batch_idx >= batch_size) return;

    // Find maximum value in the row
    float local_max = -1;   // temp in place of old const? 
    for (int i = 0; i < output_size; ++i) {
        local_max = max(local_max, input[batch_idx * output_size + i]);
    }

    // Calculate exp(input - max) and sum_exp for normalization
    float exp_sum = 0.0f;
    for (int i = 0; i < output_size; ++i) {
        exp_sum += exp(input[batch_idx * output_size + i] - local_max);
    }

    // Normalize and output the result
    output[idx] = exp(input[batch_idx * output_size + output_idx] - local_max) / exp_sum;
}


vector<float*> _fw_GPU(vector<float> X, vector<vector<float>> Ws, int n_samples, int n_features, 
                        int hidden_size, int out_size) {

    vector<float*> outs;
    outs.push_back(X.data());

    for (int i = 0; i < Ws.size(); i++) {
        if (i != 0) n_features = hidden_size;
        if (i == Ws.size() - 1) hidden_size = out_size;

        int n_inputs_per_stream = (n_samples * n_features);
        int n_outputs_per_stream = (n_samples * hidden_size);
 
        vector<float> W = Ws[i];
        float *X = outs[i];
        float *out;
        CHECK(hipHostMalloc(&out, n_outputs_per_stream * sizeof(float)));

        // Allocate memory on device
        float *d_X, *d_W, *d_out;
        CHECK(hipMalloc(&d_X, n_inputs_per_stream * sizeof(float)));
        CHECK(hipMalloc(&d_W, W.size() * sizeof(float)));
        CHECK(hipMalloc(&d_out, n_outputs_per_stream * sizeof(float)));

        // Copy memory: host-to-device
        CHECK(hipMemcpy(
            d_X, X, n_inputs_per_stream * sizeof(float), 
            hipMemcpyHostToDevice
        ));
        CHECK(hipMemcpy(d_W, W.data(), W.size() * sizeof(float), hipMemcpyHostToDevice));

        // Define block and grid size
        dim3 blockSize(16, 16);
        dim3 gridSize((hidden_size + blockSize.x - 1) / blockSize.x,
                        (n_samples + blockSize.y - 1) / blockSize.y);

        // Multiply
        _matmul_GPU<<<gridSize, blockSize>>>(d_X, d_W, d_out, n_samples, n_features, hidden_size);

        // Activation function
        dim3 blockSize_1D(256);
        dim3 gridSize_1D((n_samples * hidden_size + blockSize_1D.x - 1) / 256);
        if (i == Ws.size() - 1)
            _softmax_GPU<<<gridSize_1D, blockSize_1D>>>(d_out, d_out, n_samples, out_size);
        else
            _ReLU_GPU<<<gridSize_1D, blockSize_1D>>>(d_out, n_samples * hidden_size);

        // Copy memory: device-to-host
        CHECK(hipMemcpy(
            out, d_out, n_outputs_per_stream * sizeof(float), 
            hipMemcpyDeviceToHost
        ));

        outs.push_back(out);

        // Free device memory
        CHECK(hipFree(d_X));
        CHECK(hipFree(d_W));
        CHECK(hipFree(d_out));
    }

    return outs;
}
