#include "hip/hip_runtime.h"
#include "utils_device.h"


__global__ void _transpose_GPU(float* A, float* A_T, int n_rows, int n_cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_rows && j < n_cols) 
        A_T[n_rows * j + i] = A[n_cols * i + j];
}

__global__ void relu_derivative(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = (input[idx] >= 0) ? 1.0f : 0.0f;
    }
}

__global__ void scalar_div(float* data, int size, float scalar) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] /= scalar;
    }
}


__global__ void _add_GPU(float* A, float* B, float* C, int n, float sign) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) C[idx] = A[idx] + sign * B[idx]; 
}


__global__ void _ewmul_GPU(float* A, float* B, float* C, int n) {
    // Element-wise multiplication
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        C[idx] = A[idx] * B[idx];
    }
}


__global__ void _matmul_GPU(float* A, float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < m && col < k) {
        float value = 0;
        for (int i = 0; i < n; i++)  
            value += A[row * n + i] * B[i * k + col];
            
        C[row * k + col] = value; 
    }
}


__global__ void _tiled_matmul_GPU(float* A, float* B, float* C, int m, int n, int k) {
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float c = 0;

    for (int tile = 0; tile < (n + TILE_WIDTH)/TILE_WIDTH; tile++) {
        int tile_row = tile * TILE_WIDTH + threadIdx.y;
        int tile_col = tile * TILE_WIDTH + threadIdx.x;

        // Default tiles' values
        s_A[threadIdx.y][threadIdx.x] = 0.0;
        s_B[threadIdx.y][threadIdx.x] = 0.0;

        // Load value from inputs if in range
        if (row < m && tile_col < n) s_A[threadIdx.y][threadIdx.x] = A[row * n + tile_col];
        if (col < k && tile_row < n) s_B[threadIdx.y][threadIdx.x] = B[tile_row * k + col];
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++) 
            c += s_A[threadIdx.y][i] * s_B[i][threadIdx.x];
        __syncthreads();
    }

    if (row < m && col < k) C[row * k + col] = c;
}


__global__ void _sum_GPU(float* in, float* out, int n) {
    int numElemsBeforeBlk = blockIdx.x * blockDim.x * 2;
    int i = numElemsBeforeBlk + threadIdx.x;

    for (int stride = blockDim.x; stride > 0; stride /= 2) {
        
        if (threadIdx.x < stride) {
            if (i < n && i + stride < n)
                in[i] += in[i + stride];
        }

        __syncthreads(); // Synchronize within each block
    }
    
    if (threadIdx.x == 0)
        atomicAdd(out, in[numElemsBeforeBlk]);
}


__global__ void _ReLU_GPU(float* Z, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        Z[idx] = fmaxf(0.0f, Z[idx]);
    }
}


__global__ void _softmax_GPU(float *input, float *output, int batch_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_idx = idx / output_size;
    int output_idx = idx % output_size;

    if (batch_idx >= batch_size) return;

    // Find maximum value in the row
    float local_max = -1;
    for (int i = 0; i < output_size; ++i) {
        local_max = max(local_max, input[batch_idx * output_size + i]);
    }

    // Calculate exp(input - max) and sum_exp for normalization
    float exp_sum = 0.0f;
    for (int i = 0; i < output_size; ++i) {
        exp_sum += exp(input[batch_idx * output_size + i] - local_max);
    }

    // Normalize and output the result
    output[idx] = exp(input[batch_idx * output_size + output_idx] - local_max) / exp_sum;
}


vector<float*> _forward_GPU(float* X, vector<vector<float>> Ws, int n_samples, int n_features, 
                            int n_neurons, int out_size, bool optimize) {

    vector<float*> outs;
    outs.push_back(X);
    
    GpuTimer timer;
    float time;

    dim3 blockSize(32, 32);
    dim3 blockSize_1D(256);

    int layer_in_size = n_features;
    int layer_out_size = n_neurons;
    for (int i = 0; i < Ws.size(); i++) {
        if (i != 0) layer_in_size = n_neurons;
        if (i == Ws.size() - 1) layer_out_size = out_size;

        int n_input_elements = (n_samples * layer_in_size);
        int n_output_elements = (n_samples * layer_out_size);
 
        timer.Start();

        vector<float> W = Ws[i];
        float *X = outs[i];
        float *out;
        CHECK(hipHostMalloc(&out, n_output_elements * sizeof(float)));

        // Allocate memory on device
        float *d_X, *d_W, *d_out;
        CHECK(hipMalloc(&d_X, n_input_elements * sizeof(float)));
        CHECK(hipMalloc(&d_W, W.size() * sizeof(float)));
        CHECK(hipMalloc(&d_out, n_output_elements * sizeof(float)));

        // Copy memory: host-to-device
        CHECK(hipMemcpy(d_X, X, n_input_elements * sizeof(float), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_W, W.data(), W.size() * sizeof(float), hipMemcpyHostToDevice));

        // Define block and grid size
        dim3 gridSize((layer_out_size + blockSize.x - 1) / blockSize.x,
                        (n_samples + blockSize.y - 1) / blockSize.y);

        // Multiply
        if (optimize)
            _tiled_matmul_GPU<<<gridSize, blockSize>>>(d_X, d_W, d_out, n_samples, layer_in_size, layer_out_size);
        else
            _matmul_GPU<<<gridSize, blockSize>>>(d_X, d_W, d_out, n_samples, layer_in_size, layer_out_size);

        // Activation function
        dim3 gridSize_1D((n_samples * n_neurons + blockSize_1D.x - 1) / blockSize_1D.x);
        if (i == Ws.size() - 1)
            _softmax_GPU<<<gridSize_1D, blockSize_1D>>>(d_out, d_out, n_samples, out_size);
        else
            _ReLU_GPU<<<gridSize_1D, blockSize_1D>>>(d_out, n_samples * n_neurons);

        // Copy memory: device-to-host
        CHECK(hipMemcpy(
            out, d_out, n_output_elements * sizeof(float), 
            hipMemcpyDeviceToHost
        ));

        timer.Stop();
        time = timer.Elapsed();
        cout << "- layer " << i << " ";
        printf("forward time: %f ms\n", time);

        outs.push_back(out);

        // Free device memory
        CHECK(hipFree(d_X));
        CHECK(hipFree(d_W));
        CHECK(hipFree(d_out));
    }

    return outs;
}


vector<float*> _backward_GPU(vector<float*> outs, vector<vector<float>> Ws,
                        vector<float> y_onehot, int n_samples, int n_features,
                        int hidden_size, int n_classes) {
    vector<float*> gradients(Ws.size());

    // Fixed blocksizes
    dim3 blockSize(32, 32);
    dim3 blockSize_1D(256);

    // Final output layer error
    // delta_out = final_output - y_onehot
    float* final_output = outs.back();
    float *d_final_output, *d_y_onehot, *d_delta_out;
    CHECK(hipMalloc(&d_final_output, n_samples * n_classes * sizeof(float)));
    CHECK(hipMalloc(&d_y_onehot, n_samples * n_classes * sizeof(float)));
    CHECK(hipMalloc(&d_delta_out, n_samples * n_classes * sizeof(float)));
    
    CHECK(hipMemcpy(d_final_output, final_output, n_samples * n_classes * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y_onehot, y_onehot.data(), n_samples * n_classes * sizeof(float), hipMemcpyHostToDevice));
    dim3 gridSize_1D((n_samples * n_classes - 1) / blockSize_1D.x + 1);
    // result: d_delta_out
    _add_GPU<<<gridSize_1D, blockSize_1D>>>(d_final_output, d_y_onehot, d_delta_out, n_samples * n_classes, -1);
    float* d_delta_hidden = d_delta_out;

    // Final layer gradient
    float* final_input = outs[outs.size() - 2]; 
    float *d_final_input, *d_final_input_T, *d_grad_out;
    CHECK(hipMalloc(&d_final_input, n_samples * hidden_size * sizeof(float)));
    CHECK(hipMalloc(&d_final_input_T, hidden_size * n_samples * sizeof(float)));
    CHECK(hipMalloc(&d_grad_out, hidden_size * n_classes * sizeof(float)));

    CHECK(hipMemcpy(d_final_input, final_input, n_samples * hidden_size * sizeof(float), hipMemcpyHostToDevice));
    dim3 gridSize((hidden_size - 1) / blockSize.x + 1, 
                    (n_samples - 1) / blockSize.y + 1);
    // result: d_final_input_T
    _transpose_GPU<<<gridSize, blockSize>>>(d_final_input, d_final_input_T, n_samples, hidden_size);

    gridSize = dim3((n_classes - 1) / blockSize.x + 1, 
                    (hidden_size - 1) / blockSize.y + 1);
    // result: d_grad_out
    _matmul_GPU<<<gridSize, blockSize>>>(d_final_input_T, d_delta_out, d_grad_out, hidden_size, n_samples, n_classes);

    gridSize_1D = dim3((hidden_size * n_classes - 1) / blockSize_1D.x + 1);
    // result: d_grad_out
    scalar_div<<<gridSize_1D, blockSize_1D>>>(d_grad_out, hidden_size * n_classes, n_samples);

    gradients.back() = new float[hidden_size * n_classes];
    CHECK(hipMemcpy(gradients.back(), d_grad_out, hidden_size * n_classes * sizeof(float), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_final_input));
    CHECK(hipFree(d_final_input_T));
    CHECK(hipFree(d_grad_out));

    // BEGIN BACKPROPAGATION
    int layer_input_size = hidden_size;
    int layer_output_size = hidden_size;
    for (int layer = Ws.size() - 2; layer > -1; layer--) {
        if (layer == 0) layer_input_size = n_features;

        // Current layer input + outputs
        float* layer_input = outs[layer];
        float* layer_output = outs[layer + 1];
        float* d_layer_input, *d_layer_output;
        CHECK(hipMalloc(&d_layer_input, n_samples * layer_input_size * sizeof(float)));
        CHECK(hipMalloc(&d_layer_output, n_samples * layer_output_size * sizeof(float)));
        CHECK(hipMemcpy(d_layer_input, layer_input, n_samples * layer_input_size * sizeof(float), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_layer_output, layer_output, n_samples * layer_output_size * sizeof(float), hipMemcpyHostToDevice));

        // Obtain next layer's weights, input + output sizes
        int next_layer = layer + 1;
        int next_layer_input_size = layer_output_size;
        int next_layer_output_size = hidden_size;
        if (next_layer == Ws.size() - 1) next_layer_output_size = n_classes;

        vector<float> W_next = Ws[next_layer];
        float *d_W_next;
        CHECK(hipMalloc(&d_W_next, next_layer_input_size * next_layer_output_size * sizeof(float)));
        CHECK(hipMemcpy(d_W_next, W_next.data(), next_layer_input_size * next_layer_output_size * sizeof(float), hipMemcpyHostToDevice));
        
        // ReLU derivative
        float* dReLU;
        CHECK(hipMalloc(&dReLU, n_samples * layer_output_size * sizeof(float)));
        gridSize_1D = dim3((n_samples * layer_output_size - 1) / blockSize_1D.x + 1);
        relu_derivative<<<gridSize_1D, blockSize_1D>>>(d_layer_output, dReLU, n_samples * layer_output_size);

        // Transpose next layer's weights
        float *d_W_next_T;
        CHECK(hipMalloc(&d_W_next_T, next_layer_output_size * next_layer_input_size * sizeof(float)));
        gridSize = dim3((next_layer_output_size - 1) / blockSize.x + 1, 
                        (next_layer_input_size - 1) / blockSize.y + 1);
        _transpose_GPU<<<gridSize, blockSize>>>(d_W_next, d_W_next_T, next_layer_input_size, next_layer_output_size);
        CHECK(hipGetLastError());  // Checks for kernel errors
        CHECK(hipDeviceSynchronize());  // Ensures all operations are complete

        // Current layer's output error
        float* d_delta_hidden_temp;
        CHECK(hipMalloc(&d_delta_hidden_temp, n_samples * next_layer_input_size * sizeof(float)));
        gridSize = dim3((next_layer_input_size - 1) / blockSize.x + 1, 
                        (n_samples - 1) / blockSize.y + 1);
        _matmul_GPU<<<gridSize, blockSize>>>(d_delta_hidden, d_W_next_T, d_delta_hidden_temp, n_samples, next_layer_output_size, next_layer_input_size);
        CHECK(hipGetLastError());  // Checks for kernel errors
        CHECK(hipDeviceSynchronize());  // Ensures all operations are complete

        cout << layer << endl;
        gridSize_1D = dim3((n_samples * layer_output_size - 1) / blockSize_1D.x + 1);

        // Free old d_delta_hidden and allocate new size
        CHECK(hipFree(d_delta_hidden));
        CHECK(hipMalloc(&d_delta_hidden, n_samples * layer_output_size * sizeof(float)));

        _ewmul_GPU<<<gridSize_1D, blockSize_1D>>>(d_delta_hidden_temp, dReLU, d_delta_hidden, n_samples * layer_output_size);
        CHECK(hipGetLastError());  // Checks for kernel errors
        CHECK(hipDeviceSynchronize());  // Ensures all operations are complete

        float* d_layer_input_T;
        CHECK(hipMalloc(&d_layer_input_T, layer_input_size * n_samples * sizeof(float)));
        gridSize = dim3((layer_input_size - 1) / blockSize.x + 1, 
                        (n_samples - 1) / blockSize.y + 1);
        _transpose_GPU<<<gridSize, blockSize>>>(d_layer_input, d_layer_input_T, n_samples, layer_input_size);
        CHECK(hipGetLastError());  // Checks for kernel errors
        CHECK(hipDeviceSynchronize());  // Ensures all operations are complete
        
        // Grad hidden
        float* d_grad_hidden;
        CHECK(hipMalloc(&d_grad_hidden, layer_input_size * layer_output_size * sizeof(float)));
        
        gridSize = dim3((layer_output_size - 1) / blockSize.x + 1, 
                        (layer_input_size - 1) / blockSize.y + 1);
        _matmul_GPU<<<gridSize, blockSize>>>(d_layer_input_T, d_delta_hidden, d_grad_hidden, layer_input_size, n_samples, layer_output_size);
        gridSize_1D = dim3((layer_input_size * layer_output_size - 1) / blockSize_1D.x + 1);
        scalar_div<<<gridSize_1D, blockSize_1D>>>(d_grad_hidden, layer_input_size * layer_output_size, n_samples);

        float* grad_hidden = new float[layer_input_size * layer_output_size];
        CHECK(hipMemcpy(grad_hidden, d_grad_hidden, layer_input_size * layer_output_size * sizeof(float), hipMemcpyDeviceToHost));
        gradients[layer] = grad_hidden;

        CHECK(hipFree(d_layer_input));
        CHECK(hipFree(d_layer_output));
        CHECK(hipFree(dReLU));
        CHECK(hipFree(d_W_next));
        CHECK(hipFree(d_W_next_T));
        CHECK(hipFree(d_delta_hidden_temp));
        CHECK(hipFree(d_layer_input_T));
        CHECK(hipFree(d_grad_hidden));
    }

    CHECK(hipFree(d_final_output));
    CHECK(hipFree(d_y_onehot));
    CHECK(hipFree(d_delta_hidden));

    return gradients;
}
