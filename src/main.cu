
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <stdexcept>

using namespace std;

// Helper function to reverse byte order (endian conversion)
int ReverseInt(int i) {
    unsigned char ch1, ch2, ch3, ch4;
    ch1 = i & 255;
    ch2 = (i >> 8) & 255;
    ch3 = (i >> 16) & 255;
    ch4 = (i >> 24) & 255;
    return ((int)ch1 << 24) + ((int)ch2 << 16) + ((int)ch3 << 8) + ch4;
}

// Data loader function for images (like read_mnist_data in mnist.cc)
void readMNISTImages(const string& filename, vector<vector<float>>& images, int& numImages, int& imageSize) {
    ifstream file(filename, ios::binary);
    if (!file.is_open()) {
        throw runtime_error("Could not open file: " + filename);
    }

    int magic_number = 0;
    file.read((char*)&magic_number, sizeof(magic_number));
    magic_number = ReverseInt(magic_number);
    if (magic_number != 2051) {
        throw runtime_error("Invalid magic number in image file.");
    }

    file.read((char*)&numImages, sizeof(numImages));
    file.read((char*)&imageSize, sizeof(imageSize));
    numImages = ReverseInt(numImages);
    imageSize = ReverseInt(imageSize);

    images.resize(numImages, vector<float>(imageSize));
    
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < imageSize; j++) {
            unsigned char pixel;
            file.read((char*)&pixel, sizeof(pixel));
            images[i][j] = static_cast<float>(pixel) / 255.0f; // Normalize to [0, 1]
        }
    }

    file.close();
}

// Data loader function for labels (like read_mnist_label in mnist.cc)
void readMNISTLabels(const string& filename, vector<int>& labels, int& numLabels) {
    ifstream file(filename, ios::binary);
    if (!file.is_open()) {
        throw runtime_error("Could not open file: " + filename);
    }

    int magic_number = 0;
    file.read((char*)&magic_number, sizeof(magic_number));
    magic_number = ReverseInt(magic_number);
    if (magic_number != 2049) {
        throw runtime_error("Invalid magic number in label file.");
    }

    file.read((char*)&numLabels, sizeof(numLabels));
    numLabels = ReverseInt(numLabels);

    labels.resize(numLabels);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        file.read((char*)&label, sizeof(label));
        labels[i] = static_cast<int>(label);
    }

    file.close();
}
